#include "hip/hip_runtime.h"
/**
* @brief CUDA Math Matrix and Vector functions
* @file cuda_math.cu
* @author Benjamin Blundell <oni@section9.co.uk>
* @date 18/10/2016
*
*/


#include "cuda_math.hpp"

// CUDA Based Matrix and vector functions implementation 

__device__ float3 normalize(float3 &v) { 
  float l = 1.0f / sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
  float3 r;
  r.x = v.x * l;
  r.y = v.y * l;
  r.z = v.z * l;
  return r;
}

__device__ float3 subtract(float3 &lhs, float3 &rhs) {
  float3 r;
  r.x = lhs.x - rhs.x;
  r.y = lhs.y - rhs.y;
  r.z = lhs.z - rhs.z;
  return r;
}

__device__ float dot(float3 &lhs, float3 &rhs) {
  return lhs.x * rhs.x + lhs.y * rhs.y + lhs.z * rhs.z;
}

__device__ float3 multiply(float3 &lhs, float3 &rhs) {
  float3 r;
  r.x = lhs.x * rhs.x;
  r.y = lhs.y * rhs.y;
  r.z = lhs.z * rhs.z;
  return r;
}

__device__ float3 multiplyScalar(float3 &lhs, float rhs) {
  float3 r;
  r.x = lhs.x * rhs;
  r.y = lhs.y * rhs;
  r.z = lhs.z * rhs;
  return r;
}

__device__ float4 multiply(float4 &lhs, float4 &rhs) {
  float4 r;
  r.x = lhs.x * rhs.x;
  r.y = lhs.y * rhs.y;
  r.z = lhs.z * rhs.z;
  r.w = lhs.w * rhs.w;
  return r;
}

__device__ float4 multiplyScalar(float4 &lhs, float rhs) {
  float4 r;
  r.x = lhs.x * rhs;
  r.y = lhs.y * rhs;
  r.z = lhs.z * rhs;
  r.w = lhs.w * rhs;
  return r;
}

__device__ float3 cross(float3 &lhs, float3 &rhs) {
  float3 r;
  r.x = lhs.y*rhs.z - lhs.z*rhs.y;
  r.y = lhs.z*rhs.x - lhs.x*rhs.z;
  r.z = lhs.x*rhs.y - lhs.y*rhs.x;
  return r;
}



