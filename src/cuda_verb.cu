#include "hip/hip_runtime.h"
/**
* @brief The CUDA version of the verb
* @file cuda_verb.cu
* @author Benjamin Blundell <oni@section9.co.uk>
* @date 14/07/2016
*
*/


#include "cuda_verb.hpp"

using namespace std;
 using namespace boost::numeric;


// Our actual kernel that basically performs vector addition
__global__ void VerbSubjectZero(float *output) {

  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
  output[x]  = 0.0f;
}




// Our actual kernel that basically performs vector addition
__global__ void VerbSubjectAdd(int width, int height, float *input, float *output) {

  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

  unsigned int i = y * width + x; // index of current pixel (calculated using thread index)

  if (i < height * width){
    output[x] += input[i];   
  }
}


// Our actual kernel that runs over the screen and spits out a colour
__global__ void VerbSubjectKrn(int width, int height, float *input, float *input2, float *output) {

  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

  unsigned int i = y * width + x; // index of current pixel (calculated using thread index)
  unsigned int j= 0;

  if (i < width * height){
    output[i] = input[i] * input2[i];
  }

}


/**
 * Given a verb, return the sums of the subjects and objects
 * @param verb a string we are looking at
 * @param DICTIONARY_FAST the fast lookup dictionary
 * @param VERB_SUBJECTS the vector of vectors of verb subjects
 * @param WORD_VECTORS the word vectors converted to probabilities
 * @param BASIS_SIZE the size of our word vectors
 * @param base_vector a ublas vector of verb x verb 
 * @param add_vector a ublas vector of subjects added
 * @param krn_vector a ublas vector of verb (x) verb
 */

void read_subjects_cuda(string verb, map<string,int> & DICTIONARY_FAST,
    vector< vector<int> > & VERB_SUBJECTS,
    vector< vector<float> > & WORD_VECTORS,
    int BASIS_SIZE,
    ublas::vector<float> & base_vector,
    ublas::vector<float> & add_vector,
    ublas::vector<float> & krn_vector) {
 
  int vidx = DICTIONARY_FAST[verb];
  vector<int> subjects = VERB_SUBJECTS[vidx];

  for (int i=0; i < BASIS_SIZE; ++i){
    base_vector[i] = WORD_VECTORS[vidx][i];
  } 
 
  for (int i=0; i < BASIS_SIZE; ++i){
    add_vector[i] = 0.0f;
    
    for (int j=0; j < BASIS_SIZE; ++j){
      krn_vector[(BASIS_SIZE * i) + j] = 0.0f;
    }
  }

  for (int i : subjects) {
    ublas::vector<float> sbj_vector (BASIS_SIZE);
  
    for (int j =0; j < BASIS_SIZE; ++j) {
      sbj_vector[j] = WORD_VECTORS[i][j];
    }

    add_vector = add_vector + sbj_vector;
    ublas::vector<float> tk  = krn_mul(sbj_vector, sbj_vector);
    krn_vector = krn_vector + tk;   
  }
}


/**
 * Given a verb, return the sums of the subjects and objects
 * @param verb a string we are looking at
 * @param DICTIONARY_FAST the fast lookup dictionary
 * @param VERB_SBJ_OBJ the vector of vectors of subjects and objects
 * @param WORD_VECTORS the word vectors converted to probabilities
 * @param BASIS_SIZE the size of our word vectors
 * @param base_vector a ublas vector of verb x verb 
 * @param sum_subject a ublas vector of verb subjects summed
 * @param sum_krn a ublas vector of the verb subs objs kroneckered
 */

void read_subjects_objects_cuda(string verb, map<string,int> & DICTIONARY_FAST,
    vector< vector<int> > & VERB_SBJ_OBJ,
    vector< vector<float> > & WORD_VECTORS,
    int BASIS_SIZE,
    ublas::vector<float> & base_vector,
    ublas::vector<float> & sum_subject,
    ublas::vector<float> & sum_krn) {


  // Copy all the subject vectors into our memory block for transfer
  // into CUDA - this is likely SLOOOOW
  int vidx = DICTIONARY_FAST[verb];
  vector<int> subs_obs = VERB_SBJ_OBJ[vidx];
  vector<float> subs_obs_conv;

  // We modify the Basis size to fit it nicely into memory (ish)
  size_t real_width = BASIS_SIZE + (BASIS_SIZE % 8);

  for (int i : subs_obs) {
    for (int j = 0; j < BASIS_SIZE; ++j) {
      subs_obs_conv.push_back( WORD_VECTORS[i][j]);
    }
    for (int j = 0; j < real_width - BASIS_SIZE; ++j){
      subs_obs_conv.push_back(0);
    }
  }

  // Copy the base vector
  for (int i=0; i < BASIS_SIZE; ++i){
    base_vector[i] = WORD_VECTORS[vidx][i];
  } 
 
  cout << "Verb: " << verb << endl; 

  float* input_s;
  float* input_k;

  // Results on the device
  float* output_s;

  // results on the host
  float* result_s = new float[real_width];
  float* result_k = new float[real_width];


	// allocate memory on the CUDA device (GPU VRAM) for input, sum and kronecker
	hipMalloc(&output_s, real_width * sizeof(float));

  cout << "Cuda Error M0? " << hipGetErrorString(hipGetLastError()) << endl;
  cout << "Mem required in bytes " << subs_obs_conv.size() * sizeof(float) << endl;

  // Break up and loop

  size_t mem_size = 536870912; // Should check this against the card

  size_t ts = subs_obs_conv.size() * sizeof(float); 
  size_t num_blocks = ts / mem_size;
  size_t chunk_size = ts / num_blocks;

  if (num_blocks < 1) {
    num_blocks = 1;
    chunk_size = ts;
  }

  //size_t vecs_per_block = chunk_size / (real_width * sizeof(float));
  size_t vecs_per_block = 1;

  cout << "Sizes: " << endl;
  cout << num_blocks << ", " << vecs_per_block << ", " << chunk_size << ", " << subs_obs_conv.size() <<  endl;

  // Allocate pinned memory
  hipHostMalloc(&input_s, real_width * vecs_per_block * sizeof(float));
  hipHostMalloc(&input_k, real_width * vecs_per_block * sizeof(float));
  cout << "Cuda Error M2 " << hipGetErrorString(hipGetLastError()) << endl;

  // Zero everything first
  
  size_t idx = 0;

  dim3 threadsPerBlock(8, 1);
  dim3 numBlocks(real_width / threadsPerBlock.x, vecs_per_block);
  
  VerbSubjectZero <<< threadsPerBlock, numBlocks >>> (output_s);  

  while (idx < subs_obs_conv.size()) {
  
    size_t tsize = 1;
    //size_t tsize = vecs_per_block;
    //if (idx + vecs_per_block > subs_obs_conv.size()){
    //  tsize = subs_obs_conv.size() - idx;
    //}

    hipMemcpy( input_s, static_cast<void*>(&subs_obs_conv[idx]), tsize * real_width * sizeof(float), hipMemcpyHostToDevice);
    VerbSubjectAdd  <<< numBlocks, threadsPerBlock >>>(real_width, tsize, input_s, output_s);  

    idx +=  (tsize * real_width);
  }

  hipDeviceSynchronize(); // Probably dont need this now
  // copy results of computation from device back to host
  hipMemcpy(result_s, output_s, BASIS_SIZE * sizeof(float), hipMemcpyDeviceToHost);  
  
  for (int i =0; i < BASIS_SIZE; ++i){
    sum_subject[i] = result_s[i];  
  }


  // Now perform the kronecker which is basically an N^2 / 2 number of ops

  for (int i=0; i < BASIS_SIZE; ++i) { 

    idx = 0;

    // TODO maybe eventually use tsize here - fewer mem copies
    hipMemcpy( input_s, static_cast<void*>(&subs_obs_conv[i]), 1 * real_width * sizeof(float), hipMemcpyHostToDevice);
    
    while (idx < subs_obs_conv.size()) {
  
      size_t tsize = 1;
      //size_t tsize = vecs_per_block;
      //if (idx + vecs_per_block > subs_obs_conv.size()){
      //  tsize = subs_obs_conv.size() - idx;
      //}

      hipMemcpy( input_k, static_cast<void*>(&subs_obs_conv[idx]), tsize * real_width * sizeof(float), hipMemcpyHostToDevice);
      VerbSubjectKrn  <<< numBlocks, threadsPerBlock >>>(real_width, tsize, input_s, input_k, output_s);  

      // copy results of computation from device back to host
      hipMemcpy(result_s, output_s, real_width * sizeof(float), hipMemcpyDeviceToHost);  
  
      for (int j =0; j < BASIS_SIZE; ++j){
        sum_krn[i*BASIS_SIZE + j] += result_s[j];  
      }

      idx += (tsize * real_width);
    }

    hipDeviceSynchronize(); // Probably dont need this now
  }


  // free CUDA memory
	hipFree(output_s);  
	hipFree(input_s);
  hipFree(input_k);

  delete[] result_s;

}

/**
 * Return all the stats for all verb pairs CUDA version
 * @param VERBS_TO_CHECK a vector of VerbPair
 * @param VERB_TRANSITIVE the list of transitive verbs
 * @param VERB_INTRANSITIVE the list of intransitive verbs
 * @param BASIS_SIZE the size of our word vectors
 * @param DICTIONARY_FAST the fast dictionary 
 * @param VERB_SBJ_OBJ the vector of vectors of verb subject-object pairs
 * @param VERB_SUBJECTS the vector of verb subjects
 * @param WORD_VECTORS our word count vectors
 */


void all_count_cuda( std::vector<VerbPair> & VERBS_TO_CHECK,
  set<string> & VERB_TRANSITIVE,
  set<string> & VERB_INTRANSITIVE,
  int BASIS_SIZE,
  map<string,int> & DICTIONARY_FAST,
  vector< vector<int> > & VERB_SBJ_OBJ,
	vector< vector<int> > & VERB_SUBJECTS,
  vector< vector<float> > & WORD_VECTORS) {

	int total_verbs = 0;
	// Print out the total number we should expect
	for (int i=0; i < VERBS_TO_CHECK.size(); ++i){

		VerbPair vp = VERBS_TO_CHECK[i];
		if(VERB_TRANSITIVE.find(vp.v0) != VERB_TRANSITIVE.end() &&
			VERB_TRANSITIVE.find(vp.v1) != VERB_TRANSITIVE.end()){
			//cout << vp.v0 << "," << vp.v1 << endl;
			total_verbs ++;
		}	
	}

	cout << "Total verbs: " << s9::ToString(total_verbs) << endl; 
  cout << "verb0,verb1,base_sim,cs1,cs2,cs3,cs4,cs5,cs6,human_sim" << endl;

	int num_blocks = 1;

	//#pragma omp parallel
	//{
	//	num_blocks = omp_get_num_threads();
	//}

	int block_size = VERBS_TO_CHECK.size() / num_blocks;

  int block_id = omp_get_thread_num();
  int start = block_size * block_id;
  int end = block_size * (block_id + 1);

  if (block_id + 1 == num_blocks){
    end = VERBS_TO_CHECK.size();
  }

  ublas::vector<float> base_vector0 (BASIS_SIZE);
  ublas::vector<float> sum_subject0 (BASIS_SIZE);
  ublas::vector<float> sum_krn0 (BASIS_SIZE * BASIS_SIZE);

  ublas::vector<float> base_vector1 (BASIS_SIZE);
  ublas::vector<float> sum_subject1 (BASIS_SIZE);
  ublas::vector<float> sum_krn1 (BASIS_SIZE * BASIS_SIZE);

  ublas::vector<float> krn_base0 (BASIS_SIZE * BASIS_SIZE);
  ublas::vector<float> krn_base1 (BASIS_SIZE * BASIS_SIZE);

  for (int i=start; i < end; ++i){

    VerbPair vp = VERBS_TO_CHECK[i];

    // Set the vectors to zeros
    sum_subject0.clear();	
    sum_subject1.clear();				
    sum_krn0.clear();
    sum_krn1.clear();

    if(VERB_TRANSITIVE.find(vp.v0) != VERB_TRANSITIVE.end() &&
        VERB_TRANSITIVE.find(vp.v1) != VERB_TRANSITIVE.end()){

      read_subjects_objects_cuda(vp.v0,DICTIONARY_FAST, VERB_SBJ_OBJ, WORD_VECTORS, BASIS_SIZE, base_vector0, sum_subject0, sum_krn0);

      read_subjects_objects_cuda(vp.v1,DICTIONARY_FAST, VERB_SBJ_OBJ, WORD_VECTORS, BASIS_SIZE, base_vector1, sum_subject1, sum_krn1);
 
      ublas::vector<float> tv0 (BASIS_SIZE);
      ublas::vector<float> tv1 (BASIS_SIZE);

      tv0 = sum_subject0 + base_vector0;
      tv1 = sum_subject1 + base_vector1;

      float c2 = cosine_sim(tv0,tv1);
      cout << "CUDA cosine sim " << c2 << endl;

      krn_base0 = krn_mul(base_vector0, base_vector0);
      krn_base1 = krn_mul(base_vector1, base_vector1);
   
      float c4 = cosine_sim(sum_krn0, sum_krn1);

      cout << "CUDA cosine sim krn " << c4 << endl;
      //break;

    } else if(VERB_TRANSITIVE.find(vp.v0) == VERB_TRANSITIVE.end() &&
        VERB_TRANSITIVE.find(vp.v1) != VERB_TRANSITIVE.end()){

      //read_subjects_cuda(vp.v0,DICTIONARY_FAST, VERB_SUBJECTS, WORD_VECTORS, BASIS_SIZE, base_vector0, sum_subject0, sum_krn0);

      //read_subjects_objects_cuda(vp.v1,DICTIONARY_FAST, VERB_SBJ_OBJ, WORD_VECTORS, BASIS_SIZE, base_vector1, sum_subject1, sum_krn1);

      continue;

    } else if(VERB_TRANSITIVE.find(vp.v0) != VERB_TRANSITIVE.end() &&
        VERB_TRANSITIVE.find(vp.v1) == VERB_TRANSITIVE.end()){
    
      //read_subjects_objects_cuda(vp.v0,DICTIONARY_FAST, VERB_SBJ_OBJ, WORD_VECTORS, BASIS_SIZE, base_vector0, sum_subject0, sum_krn0);

      //read_subjects_cuda(vp.v1,DICTIONARY_FAST, VERB_SUBJECTS, WORD_VECTORS, BASIS_SIZE, base_vector1, sum_subject1, sum_krn1);
      continue;

    } else {
    
      //read_subjects_cuda(vp.v0,DICTIONARY_FAST, VERB_SUBJECTS, WORD_VECTORS, BASIS_SIZE, base_vector0, sum_subject0, sum_krn0);

      //read_subjects_cuda(vp.v1,DICTIONARY_FAST, VERB_SUBJECTS, WORD_VECTORS, BASIS_SIZE, base_vector1, sum_subject1, sum_krn1);
      continue;
    }

    /*krn_base0 = krn_mul(base_vector0, base_vector0);
    krn_base1 = krn_mul(base_vector1, base_vector1);
    
    float c0 = cosine_sim(base_vector0, base_vector1);
    float c1 = cosine_sim(sum_subject0, sum_subject1);
    
    ublas::vector<float> tv0 (BASIS_SIZE);
    ublas::vector<float> tv1 (BASIS_SIZE);

    tv0 = sum_subject0 + base_vector0;
    tv1 = sum_subject1 + base_vector1;

    float c2 = cosine_sim(tv0,tv1);
    tv0 = mul_vec(sum_subject0,base_vector0);
    tv1 = mul_vec(sum_subject1,base_vector1);
    float c3 = cosine_sim(tv0,tv1);
    float c4 = cosine_sim(sum_krn0, sum_krn1);


    ublas::vector<float> tk0 (BASIS_SIZE * BASIS_SIZE);
    tk0 = sum_krn0 + krn_base0;

    ublas::vector<float> tk1 (BASIS_SIZE * BASIS_SIZE);
    tk1 = sum_krn1 + krn_base1;
    float c5 = cosine_sim(tk0, tk1);

    tk0 = mul_vec(sum_krn0, krn_base0);
    tk1 = mul_vec(sum_krn1,krn_base1);
    float c6 = cosine_sim(tk0, tk1);

    std::stringstream stream;

    stream << vp.v0 << "," << vp.v1 << "," << s9::ToString(c0)
      << "," << s9::ToString(c1)
      << "," << s9::ToString(c2)
      << "," << s9::ToString(c3)
      << "," << s9::ToString(c4)
      << "," << s9::ToString(c5)
      << "," << s9::ToString(c6)
      << "," << s9::ToString(vp.s)
      << endl;

    std::cout << stream.str();
*/
  }
}
